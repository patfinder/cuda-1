
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;
    float *d_a;

    a = (float*)malloc(sizeof(float) * N);

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add<<<1,1>>>(out, d_a, b, N);

    // Cleanup after kernel execution
    hipFree(d_a);
    free(a);
}

